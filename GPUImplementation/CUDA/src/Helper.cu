#ifndef _IMPORTS
#define _IMPORTS

#include "../headers/imports.h"

#endif

using namespace std;
hipEvent_t start, stop;

void handleCudaError(int error) {
    switch (error) {
        case hipSuccess:
            return;
        case hipErrorInvalidValue:
            printf("Cuda Error: Invalide Value, have you created all events?\n");
            break;
        case hipErrorNotInitialized:
            printf("Cuda Error: Initialization Error\n");
            break;
        case hipErrorPriorLaunchFailure:
            printf("Cuda Error: Prior Launch Failure, An error has appeared somewhere :/\n");
            break;
        case hipErrorInvalidHandle:
            printf("Cuda Error: Invalide Resource Handle\n");
            break;
        default:
            printf("ERROR NOT RECOGNIZED:%s\n", hipGetErrorString((hipError_t) error));
    }
    exit(-1);
}

void startStopWatch () {
	handleCudaError(hipEventCreate(&start));
	handleCudaError(hipEventCreate(&stop));
	handleCudaError(hipEventRecord(start));
}

float stopStopWatch () {
	handleCudaError(hipEventRecord(stop));
	handleCudaError(hipEventSynchronize(stop));
	float time = 0;
    handleCudaError(hipEventElapsedTime(&time, start, stop));
    return time;
}

void printMatrix(real* matrix,int x,int y){
	for(int i=0;i<y;i++){
		for(int j=0;j<x;j++){
			if(matrix[i*x+j]<0.001) printf("0 ");
			else printf("%f ",matrix[i*x+j]);
		}
		printf("\n");
	}
}
 
